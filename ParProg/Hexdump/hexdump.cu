
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <unistd.h>
//#include <stdlib.h>

const long N = 1 << 28;
char x[N];
char y[4*N];

__global__
void hexdump(char *y, char *x, size_t n)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    char lowest = x[i] & 15;
    char highest = x[i] >> 4;
    y[4*i] = '\\';
    y[4*i+1] = 'x';
    y[4*i+2] = highest + (highest > 10 ? 'a' - 10 : '0');
    y[4*i+3] = lowest + (lowest > 10 ? 'a' - 10 : '0');
  }
}

int main(int argc, char * argv[])
{
  if (argc < 2)
  {
    printf("Usage: %s <filename>\n", argv[0]);
    return 1;
  }
  char *d_x, *d_y;
  FILE * binaryFile = fopen(argv[1], "rb");
  //x = (char*)calloc(N, sizeof(char));
  //y = (char*)calloc(4*N, sizeof(char));
  long n = fread(x, sizeof(char), N, binaryFile);
  hipMalloc(&d_x, n*sizeof(char));
  hipMalloc(&d_y, 4*n*sizeof(char));


  hipMemcpy(d_x, x, n*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, 4*n*sizeof(char), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  //saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
  //sleep(20);
  hexdump<<<(N+255)/256, 256>>>(d_y, d_x, n);
  hipMemcpy(y, d_y, 4*n*sizeof(char), hipMemcpyDeviceToHost);
  //sleep(30);
  y[4*n-1] = 0;
  printf("%s", y);
  hipFree(d_x);
  hipFree(d_y);
  //free(x);
  //free(y);
 }
